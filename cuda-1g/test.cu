
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(){
	int num;
	hipGetDeviceCount(&num);
	printf("%d gpus available\n", num);
	
	for (int i = 0; i < num; i++)
	{
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device %d has compute capability %d %d\n", prop.maxBlocksPerMultiProcessor, prop.maxThreadsPerMultiProcessor, prop.multiProcessorCount);
	}
	
}

